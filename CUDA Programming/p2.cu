#include <stdio.h>
#include <stdlib.h>
#include <math.h>
#include <hip/hip_runtime.h>

/* first grid point */
#define   XI              0.0
/* last grid point */
#define   XF              M_PI


/* function declarations */
//double     fn(double);
void        print_function_data(int, double*, double*, double*);
int         main(int, char**);

// cos function in the device 
__device__ double cos_func( double  a )         
{
        return cos(a);
}
// Kernel definition with __global__ keyword
__global__ void integral_func(int N, double *x, double *y, double *inf_arr, double *d_area, double h1)
{       
        // To identify each thread uniquely
        int i = blockIdx.x * blockDim.x + threadIdx.x;         
        
        // To restrict the threads that need to function and to avoid looking into out of bounds memory spaces
        if (i < N)                                      
        {
                y[i] = cos_func(x[i]); // Each thread will perform function call on its respective i element of the xc array
        }
        __syncthreads();  // So that y array is filled before any race between threads occur
        
        if(i < N && i!=0)               
        {
                d_area[i] = (y[i] + y[i-1])/2*h1; // For all threads except thread 0 since y[0-1] since y[-1] is out of bounds
        }
        
        else if(i == 0 )  // have to specify i == 0 and so that threads greater than N don't start executing 
        {       
                d_area[i] = (y[i] + cos_func(0))/2*h1;
        }
        __syncthreads(); 

        //if(i < N && i!=0)
        if(i < N) // Copying d_area values into inf_arr
        {       
                inf_arr[i] = d_area[i];         
        }
        // else if(i == 0)
        // {
        //         inf_arr[0] = d_area[0];
        // }
        __syncthreads();
        if(i < N && i!=0) 
        {
                for(int j = i; j > 0; j--) // Each thread computes from its respective index to 0 the cumulative sum and puts it in inf_arr[i]
                {
                        inf_arr[i] = inf_arr[i] + d_area[j-1];  
                } 
        }
}

int main (int argc, char *argv[])
{
        int NGRID;
        if(argc > 1)
            NGRID = atoi(argv[1]);
        else 
        {
                printf("Please specify the number of grid points.\n");
                exit(0);
        }
        //loop index
        int     i;
        double  h;

        double *inf = (double *)malloc(sizeof(double) * (NGRID) );
        double  *xc = (double *)malloc(sizeof(double)* (NGRID + 1));
        double  *yc = (double*)malloc(sizeof(double) * (NGRID));
        double *new_xc =(double*) malloc((NGRID) * sizeof(double));

        // GPU variables memory allocation
        double *d_xc, *d_yc, *d_inf, *d_area;
        hipMalloc(&d_xc, NGRID*sizeof(double));
        hipMalloc(&d_yc, NGRID*sizeof(double));
        hipMalloc(&d_inf, NGRID*sizeof(double));
        //cudaMalloc(&d_inf, NGRID*sizeof(double));
        hipMalloc(&d_area, NGRID*sizeof(double));

        //construct grid of x axis
        for (i = 1; i <= NGRID ; i++)
        {
                xc[i] = XI + (XF - XI) * (double)(i - 1)/(double)(NGRID - 1);
        }

        // To remove the extra index before passing it to the device
        for(i = 0; i < NGRID; i++)
        {
                new_xc[i] = xc[i+1];
        }
        //int  imin, imax;  
        //imin = 1;
        //imax = NGRID;
        // get the y value of the origin function, yc array is used for output
        // should not use for computing on GPU
        //for( i = imin; i <= imax; i++ )
        //{
        //        yc[i] = fn(xc[i]);
        //}

        // Deep copy of host array to be passed to device variable
        hipMemcpy(d_xc, new_xc , NGRID*sizeof(double), hipMemcpyHostToDevice); // To copy memory of alloted space into device global memory 

        //inf[0] = 0.0;
        h = (XF - XI) / (NGRID - 1);

        // Computing execution configuration
        int numOfThreads = 512;         // Setting maximum number of values
        int numOfBlocks = NGRID/numOfThreads + (NGRID%numOfThreads == 0? 0:1);  // Dynamically determining the number of blocks.

        // Kernel function call
        integral_func<<<numOfBlocks, numOfThreads>>>(NGRID, d_xc, d_yc, d_inf, d_area, h);      
        
        // Copying data back to host
        hipMemcpy(yc, d_yc, NGRID*sizeof(double), hipMemcpyDeviceToHost); 
        hipMemcpy(inf, d_inf, NGRID*sizeof(double), hipMemcpyDeviceToHost);
        
        // you should parallel the following computation workload on GPU
        //for(i = 1 ; i <= NGRID; ++i){
        //    area = ( fn(xc[i]) + fn(xc[i-1]) ) / 2 * h; 
        //    inf[i] = inf[i-1] + area;
        //}
        
        print_function_data(NGRID, &xc[1], &yc[0], &inf[0]);

        //free allocated memory 
        hipFree(d_xc);
        hipFree(d_yc);
        hipFree(d_inf);
        free(xc);
        free(yc);
        free(inf);

        return 0;
}

//prints out the function and its derivative to a file
void print_function_data(int np, double *x, double *y, double *dydx)
{
        int   i;

        char filename[1024];
        sprintf(filename, "fn-%d.dat", np);

        FILE *fp = fopen(filename, "w");

        for(i = 0; i < np; i++)
        {
                fprintf(fp, "%f %f %f\n", x[i], y[i], dydx[i]);
        }

        fclose(fp);
}
